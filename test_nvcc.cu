// Copyright (c) 2019, NVIDIA Corporation. All rights reserved.
//
// This work is made available under the Nvidia Source Code License-NC.
// To view a copy of this license, visit
// https://nvlabs.github.io/stylegan2/license.html


#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void cudaKernel(void)
{
    printf("GPU says hello!\n");
}

int main(void)
{
    printf("CPU says hello!\n");
    hipError_t err = hipLaunchKernel(reinterpret_cast<const void*>(cudaKernel), 1, 1, NULL, 0, NULL);
    if (err != hipSuccess)
    {
        printf("%s: %s\n", hipGetErrorName(err), hipGetErrorString(err));
        return 1;
    }
    return 0;
}
